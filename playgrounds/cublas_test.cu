
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <iostream>

int main() {
    hipblasHandle_t handle;
    if (hipblasCreate(&handle) != HIPBLAS_STATUS_SUCCESS) {
        std::cerr << "cuBLAS initialization failed!" << std::endl;
        return 1;
    }
    std::cout << "cuBLAS initialized successfully!" << std::endl;
    hipblasDestroy(handle);
    return 0;
}
