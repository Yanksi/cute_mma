#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2023 - 2024 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/
#include <cstdlib>
#include <cstdio>
#include <cassert>
#include <random>
#include "cute_oft_simple.hpp"
#ifdef USE_CUBLAS
#include "cublas_oft.hpp"
#endif

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <cute/tensor.hpp>
#include <cute/atom/mma_atom.hpp>

#include <cutlass/util/print_error.hpp>
#include <cutlass/util/GPU_Clock.hpp>
#include <cutlass/util/helper_cuda.hpp>

#include <argparse/argparse.hpp>
#include <vector>

#define mmax(a,b) ((a) > (b) ? (a) : (b))
#define mmin(a,b) ((a) < (b) ? (a) : (b))

#ifdef DEBUG
#define GROUP_SIZE 64
#else
#define GROUP_SIZE 256
#endif

namespace cute {
  template <typename TO, typename TR>
  struct Params {
    static_assert(sizeof(TO) == 0, "This struct should not be used");
  };

  template <>
  struct Params <half, half> {
    static const unsigned int bM = 128;
    static const unsigned int bN_group = 1;
    static const unsigned int bK_block = 2;
    static const unsigned int bP = 3;
    static const bool block_tiling_copy = true;
    using warp_layout = Layout<Shape<Int<4>, Int<2>>>;
    // using mma_atom = SM80_16x8x8_F16F16F16F16_TN;
    // using s2r_atom = Copy_Atom<SM75_U32x4_LDSM_N, half_t>;
  };
}

template <typename copy_as_t, typename ele_t, bool k_major, bool block_tiling,
  typename _BM, typename _BK, typename _N_Threads>
constexpr auto cp_layout(_BM bm, _BK bk, _N_Threads _total_threads) {
  using namespace cute;
  constexpr int vec_width = sizeof(copy_as_t) / sizeof(ele_t);
  constexpr int total_elements = bm * bk;

  constexpr int needed_threads = total_elements / vec_width;
  CUTE_STATIC_ASSERT(total_elements % vec_width == 0, "total number of elements shall be divisible by the vector length");
  constexpr int total_threads = mmin(_total_threads, needed_threads);

  constexpr int elements_per_thread = total_elements / total_threads;
  CUTE_STATIC_ASSERT(total_elements % total_threads == 0, "total number of elements shall be divisible by the number of threads using");
  CUTE_STATIC_ASSERT(elements_per_thread % vec_width == 0, "number of elements handled by each thread should be divisible by the vector width");
  constexpr int cp_width = (block_tiling) ? vec_width : elements_per_thread;
  if constexpr (k_major) {
    CUTE_STATIC_ASSERT(!block_tiling || bk % cp_width == 0);
    CUTE_STATIC_ASSERT(block_tiling || (bk % cp_width == 0 || cp_width % bk == 0));
    constexpr int threads_along_k = mmax(bk / cp_width, 1);
    constexpr int threads_k_size = bk / threads_along_k;
    constexpr int threads_m_size = mmax(cp_width / bk, 1);
    constexpr int threads_along_m = total_threads / threads_along_k;
    return make_tiled_copy(Copy_Atom<SM80_CP_ASYNC_CACHEALWAYS<copy_as_t>, ele_t>{},
                           make_layout(Shape<Int<threads_along_m>, Int<threads_along_k>>{}, LayoutRight{}),
                          //  Layout<Shape<Int<threads_along_m>, Int<threads_along_k>>>{},
                           Layout<Shape<Int<threads_m_size>, Int<threads_k_size>>>{});
  } else {
    // As it not really possible to have copy width greater than bm, we don't need to check for that
    CUTE_STATIC_ASSERT(bm % cp_width == 0);
    constexpr int threads_along_m = bm / cp_width;
    constexpr int threads_along_k = total_threads / threads_along_m;
    // return make_tiled_copy(Copy_Atom<UniversalCopy<copy_as_t>, ele_t>{},
    return make_tiled_copy(Copy_Atom<SM80_CP_ASYNC_CACHEALWAYS<copy_as_t>, ele_t>{},
                           Layout<Shape<Int<threads_along_m>, Int<threads_along_k>>>{},
                           Layout<Shape<Int<cp_width>, _1>>{});
  }
}

// // Setup params for a TN GEMM, K-Major inputs
void oft_tn(int m, int n, int k,
        half const* A, int ldA,
        half const* B, int ldB,
        half const* R, int ldR,
        half      * C, int ldC,
        hipStream_t stream = 0)
{
  using namespace cute;

  // Define shapes (dynamic)
  auto M = int(m);
  auto N = int(n);
  auto K = int(k);
  auto prob_shape = make_shape(M, N, K);                     // (M, N, K)

  using CurrParams = Params<half, half>;

  // Define CTA tile sizes (static)
  auto group_size = Int<GROUP_SIZE>{}; // Group size for the block tiling
  auto reconn_sz = _8{}; // hardcoded for now, can be made dynamic later
  auto bM = Int<CurrParams::bM>{};
  auto bN_group = Int<CurrParams::bN_group>{};
  auto bN = bN_group * group_size;
  auto bK_block = Int<CurrParams::bK_block>{};
  auto bK = bK_block * reconn_sz;
  auto blocks_tiler = make_shape(bM, bN_group, bK_block);                   // (BLK_M, BLK_N, BLK_K)
  auto bP = Int<CurrParams::bP>{};  // Pipeline
  int n_groups = N / group_size;
  auto warp_layout = typename CurrParams::warp_layout{};

  // Define the gmem layouts
  auto A_layout = make_layout(
    make_shape(M, K),
    make_stride(ldA, Int<1>{})
  );

  auto B_layout = make_layout(
    make_shape(N, K),
    make_stride(ldB, Int<1>{})
  );

  auto R_layout = make_layout(
    make_shape(n_groups * reconn_sz, K),
    make_stride(ldR, Int<1>{})
  );

  auto C_layout = make_layout(
    make_shape(M, N),
    make_stride(ldC, Int<1>{})
  );

  TiledCopy copyA = cp_layout<uint128_t, half, true, CurrParams::block_tiling_copy>(bM, bK, size(warp_layout) * _32{});
  TiledCopy copyB = cp_layout<uint128_t, half, true, CurrParams::block_tiling_copy>(bN, bK, size(warp_layout) * _32{});
  TiledCopy copyR = cp_layout<uint128_t, half, true, CurrParams::block_tiling_copy>(bN_group * reconn_sz, bK, size(warp_layout) * _32{});

  dim3 dimBlock(size(warp_layout) * _32{});
  dim3 dimGrid(size(ceil_div(M, bM)),
               size(ceil_div(N, bN)));
  oft_device<<<dimGrid, dimBlock, 0, stream>>>
      (prob_shape, blocks_tiler,
       A, A_layout, copyA,
       R, R_layout, copyR, group_size, reconn_sz,
       B, B_layout, copyB,
       C, C_layout, warp_layout, bP);
}

int main(int argc, char** argv)
{
  using namespace cute;
  argparse::ArgumentParser program(std::string("oft"));
  program.add_argument("-m", "--m")
    .help("Number of rows in matrix A")
    .default_value(128)
    .action([](const std::string& value) { return std::stoi(value); });
  program.add_argument("-n", "--n")
    .help("Number of columns in matrix B")
    .default_value(64)
    .action([](const std::string& value) { return std::stoi(value); });
  program.add_argument("-k", "--k")
    .help("Number of columns in matrix A and rows in matrix B")
    .default_value(32)
    .action([](const std::string& value) { return std::stoi(value); });
  program.add_argument("-t", "--timing_iterations")
    .help("Number of iterations to time")
    .default_value(100)
    .action([](const std::string& value) { return std::stoi(value); });

  #ifdef DEBUG
  program.add_argument("-p", "--print_matrices")
    .help("Print matrices A, B, R")
    .default_value(false)
    .implicit_value(true);
  #endif

  #ifdef USE_CUBLAS
  program.add_argument("--cublas_mode")
    .help("The mode for the cublas kernel, either 'AR_W' or 'A_RW'")
    .default_value(std::string(""))
    .action([](const std::string& value) { return value; });
  #endif
  
  try {
    program.parse_args(argc, argv);
  } catch (const std::runtime_error& err) {
    std::cout << err.what() << std::endl;
    std::cout << program;
    return 1;
  }
  
  int m = program.get<int>("--m");
  int n = program.get<int>("--n");
  int k = program.get<int>("--k");
  int timing_iterations = program.get<int>("--timing_iterations");

  #ifdef USE_CUBLAS
  std::string cublas_mode = program.get<std::string>("--cublas_mode");
  #endif

  std::cout << "M = " << m << std::endl;
  std::cout << "N = " << n << std::endl;
  std::cout << "K = " << k << std::endl;

  int n_groups = n / GROUP_SIZE;

  thrust::host_vector<half> h_A(m * k);
  thrust::host_vector<half> h_B(n * k);
  thrust::host_vector<half> h_R(n_groups * 8 * k); // 8 is the hardcoded reconnection size
  thrust::host_vector<half> h_C(m * n);

  Tensor h_A_tensor = make_tensor(h_A.data(), make_shape(m, k), LayoutRight{});
  Tensor h_B_tensor = make_tensor(h_B.data(), make_shape(n, k), LayoutRight{});
  Tensor h_R_tensor = make_tensor(h_R.data(), make_shape(n_groups * 8, k), LayoutRight{});
  Tensor h_R_4d = zipped_divide(
    h_R_tensor,
    make_tile(
      make_layout(8), // hardcoded reconnection size
      make_layout(8)  // hardcoded reconnection size
    )
  );

  for (int i = 0; i < size<0>(h_A_tensor); ++i) {
    for (int j = 0; j < size<1>(h_A_tensor); ++j) {
      h_A_tensor(i, j) = static_cast<half>( (rand() / double(RAND_MAX)) );
    }
  }

  for (int i = 0; i < size<0>(h_B_tensor); ++i) {
    for (int j = 0; j < size<1>(h_B_tensor); ++j) {
      h_B_tensor(i, j) = static_cast<half>( (rand() / double(RAND_MAX)) );
    }
  }
  
  int shuffle_idx[8] = {0, 1, 2, 3, 4, 5, 6, 7};
  for (int i = 0; i < size<1>(h_R_4d); ++i) {
    std::shuffle(std::begin(shuffle_idx), std::end(shuffle_idx), std::mt19937{std::random_device{}()});
    for (int j = 0; j < 8; ++j) { // hardcoded reconnection size
      // shuffle the indices to create a more complex pattern
      h_R_4d(make_coord(j, shuffle_idx[j]), i) = static_cast<half>(1.0f);
    }
  }

  for (int j = 0; j < m*n; ++j) h_C[j] = static_cast<half>(-1);

  #ifdef DEBUG
  if (program.get<bool>("--print_matrices")) {
    printf("A:\n");
    for (int i = 0; i < size<0>(h_A_tensor); ++i) {
      for (int j = 0; j < size<1>(h_A_tensor); ++j) {
        printf("%6.3f ", static_cast<float>(h_A_tensor(i, j)));
      }
      printf("\n");
    }

    printf("R:\n");
    for (int i = 0; i < size<0>(h_R_tensor); ++i) {
      for (int j = 0; j < size<1>(h_R_tensor); ++j) {
        printf("%6.3f ", static_cast<float>(h_R_tensor(i, j)));
      }
      printf("\n");
    }

    printf("B:\n");
    for (int i = 0; i < size<0>(h_B_tensor); ++i) {
      for (int j = 0; j < size<1>(h_B_tensor); ++j) {
        printf("%6.3f ", static_cast<float>(h_B_tensor(i, j)));
      }
      printf("\n");
    }
  }
  #endif

  thrust::device_vector<half> d_A = h_A;
  thrust::device_vector<half> d_B = h_B;
  thrust::device_vector<half> d_C = h_C;
  thrust::device_vector<half> d_R = h_R;

  std::vector<std::function<void()>> test_funcs;
  test_funcs.push_back([&]() {
    oft_tn(m, n, k,
      d_A.data().get(), k,
      d_B.data().get(), k,
      d_R.data().get(), k,
      d_C.data().get(), n);
  });

  #ifdef USE_CUBLAS
  hipblasHandle_t cublas_handle;
  getCublasTensorOpHandle(&cublas_handle);
  test_funcs.push_back([&]() {
    cublas_oft(d_A, d_R, d_B, d_C, m, GROUP_SIZE, n_groups, k, 8, &cublas_handle, false);
    GEMM_CHECK_CUDA(hipDeviceSynchronize());
  });
  test_funcs.push_back([&]() {
    cublas_oft(d_A, d_R, d_B, d_C, m, GROUP_SIZE, n_groups, k, 8, &cublas_handle, true);
    GEMM_CHECK_CUDA(hipDeviceSynchronize());
  });
  #endif

  #ifdef DEBUG
  test_funcs[0](); // warmup
  CUTE_CHECK_LAST();
  thrust::host_vector<half> h_C_result = d_C;
  d_C.assign(h_C.begin(), h_C.end()); // reset d_C to initial state
  test_funcs[1](); // warmup
  thrust::host_vector<half> h_C_ref = d_C;
  bool check_result = true;
  for (int i = 0; i < h_C_result.size(); ++i) {
    float ref_val = static_cast<float>(h_C_ref[i]);
    float result_val = static_cast<float>(h_C_result[i]);
    if (abs((ref_val - result_val) / ref_val)  > 5e-3f) {
      printf("Mismatch at index %d: %f != %f\n", i, static_cast<float>(h_C_result[i]), static_cast<float>(h_C_ref[i]));
      check_result = false;
      // return 1;
    }
  }
  if (check_result) {
    std::cout << "All results match!" << std::endl;
  } else {
    std::cout << "Some results do not match!" << std::endl;
  }
  #else

  double n_blocks = k / 8.0; // hardcoded reconnection size
  double base_t_flops = (double)m * n * k * 2.0 * 1e-12; // 2 flops per multiply-add
  printf("Base TFLOPS: %.5f\n", base_t_flops);
  double t_flops_AR_W = (((double)n_groups * m * k * k) / n_blocks) * 2.0 * 1e-12 + base_t_flops; // 2 flops per multiply-add
  double t_flops_AR_W_sparse = (((double)n_groups * m * k * k) / n_blocks) * 2.0 * 1e-12 + base_t_flops * 2; // 2 flops per multiply-add
  double t_flops_A_RW = (((double)n * k * k) / n_blocks) * 2.0 * 1e-12 + base_t_flops; // 2 flops per multiply-add
  printf("Total TFLOPS (AR_W): %.5f, (AR_W_sparse): %.5f, (A_RW): %.5f\n", t_flops_AR_W, t_flops_AR_W_sparse, t_flops_A_RW);

  auto test_func = test_funcs[0];
  #ifdef USE_CUBLAS
  if (cublas_mode == "AR_W") {
    test_func = test_funcs[1];
  } else if (cublas_mode == "A_RW") {
    test_func = test_funcs[2];
  }
  #endif
  test_func(); // warmup
  CUTE_CHECK_LAST();

  // Timing iterations
  GPU_Clock timer;
  timer.start();
  for (int i = 0; i < timing_iterations; ++i) {
    test_func();
  }
  double time = timer.seconds() / timing_iterations;
  CUTE_CHECK_LAST();
  printf("TFLOPS/s (AR_W): %.2f, (AR_W_sparse): %.2f, (A_RW): %.2f, Time: %.3f ms\n",
         t_flops_AR_W / time, t_flops_AR_W_sparse / time, t_flops_A_RW / time, time * 1000.0);
  #endif
  return 0;
}
